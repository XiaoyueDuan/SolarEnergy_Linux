#include "hip/hip_runtime.h"
#include "FocusFlatRectangleHelio.cuh"

void FocusFlatRectangleHelio::setSize(float3 size) {
    size_ = size;
}

namespace focusFlatRectangle_heliostat {
    // Step 1: Generate local micro-heliostats' centers and normals
    __global__ void mapMicrohelioCentersAndNormal(float3 *d_microhelio_centers, float3 *d_microhelio_normals,
                                                  float3 *d_local_normal, float3 *d_local_centers,
                                                  float3 subhelio_size,
                                                  const int2 row_col, const int2 sub_row_col,
                                                  const float pixel_length, const size_t size) {
        int myId = global_func::getThreadId();
        if (myId >= size)
            return;

        int row = myId / (row_col.y * sub_row_col.y);
        int col = myId % (row_col.y * sub_row_col.y);
        int block_id = global_func::unroll_index(make_int2(row / sub_row_col.x, col / sub_row_col.y), row_col);
        // 1. Normal
        d_microhelio_normals[myId] = d_local_normal[block_id];

        // 2. Centers
        row %= sub_row_col.x;
        col %= sub_row_col.y;
        // 2.1 Rotate
        float3 local_pos = make_float3((col + 0.5f) * pixel_length - subhelio_size.x / 2,
                                       0.0f,
                                       (row + 0.5f) * pixel_length - subhelio_size.z / 2);
        local_pos = focusFlatRectangleHeliostatLocal2World(local_pos, d_local_normal[block_id]);

        // 2.2 Transform
        d_microhelio_centers[myId] = global_func::transform(local_pos, d_local_centers[block_id]);
    }

    // Step 2: Generate micro-heliostats' normals
    __global__ void map_microhelio_normals(float3 *d_microhelio_world_normals,
                                           float3 *d_microhelio_local_normals, float3 normal, const size_t size) {
        int myId = global_func::getThreadId();
        if (myId >= size)
            return;

        d_microhelio_world_normals[myId] = global_func::local2world(d_microhelio_local_normals[myId], normal);

    }

    // Step 3: Transform local micro-helio center to world postion
    __global__ void map_microhelio_center2world(float3 *d_microhelio_world_centers, float3 *d_microhelio_local_centers,
                                                const float3 normal, const float3 world_pos, const size_t size) {
        int myId = global_func::getThreadId();
        if (myId >= size)
            return;

        float3 local = d_microhelio_local_centers[myId];
        local = focusFlatRectangleHeliostatLocal2World(local, normal);    // Then Rotate
        local = global_func::transform(local, world_pos);   // Translation to the world system
        d_microhelio_world_centers[myId] = local;
    }
}

int FocusFlatRectangleHelio::CGetDiscreteMicroHelioOriginsAndNormals(float3 *&d_microhelio_centers,
                                                                     float3 *&d_microhelio_normals) {
    float2 subhelio_row_col_length;
    subhelio_row_col_length.x = (size_.z - gap_.y * (row_col_.x - 1)) / float(row_col_.x);
    subhelio_row_col_length.y = (size_.x - gap_.x * (row_col_.y - 1)) / float(row_col_.y);

    int2 sub_row_col;
    sub_row_col.x = subhelio_row_col_length.x / pixel_length_;
    sub_row_col.y = subhelio_row_col_length.y / pixel_length_;

    int map_size = sub_row_col.x * sub_row_col.y * row_col_.x * row_col_.y;

    int nThreads;
    dim3 nBlocks;
    global_func::setThreadsBlocks(nBlocks, nThreads, map_size);

    // 1. local center position
    if (d_microhelio_centers == nullptr)
        checkCudaErrors(hipMalloc((void **) &d_microhelio_centers, sizeof(float3) * map_size));
    if (d_microhelio_normals == nullptr)
        checkCudaErrors(hipMalloc((void **) &d_microhelio_normals, sizeof(float3) * map_size));
    focusFlatRectangle_heliostat::
    mapMicrohelioCentersAndNormal << < nBlocks, nThreads >> > (d_microhelio_centers, d_microhelio_normals,
            d_local_normals, d_local_centers,
            make_float3(subhelio_row_col_length.y, size_.y, subhelio_row_col_length.x),
            row_col_, sub_row_col, pixel_length_, map_size);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

    // 2. normal
    focusFlatRectangle_heliostat::
    map_microhelio_normals << < nBlocks, nThreads >> > (d_microhelio_normals, d_microhelio_normals, normal_, map_size);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

    // 3. world center position
    focusFlatRectangle_heliostat::
        map_microhelio_center2world << < nBlocks, nThreads >> >(d_microhelio_centers, d_microhelio_centers, normal_, pos_, map_size);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

    return map_size;
}

void FocusFlatRectangleHelio::CGetSubHeliostatVertexes(std::vector<float3> &subHeliostatVertexes) {
    float2 subhelio_row_col_length;
    subhelio_row_col_length.x = (size_.z - gap_.y * (row_col_.x - 1)) / float(row_col_.x);
    subhelio_row_col_length.y = (size_.x - gap_.x * (row_col_.y - 1)) / float(row_col_.y);

    std::vector<float3> localSubVertexes;
    localSubVertexes.push_back(make_float3(-subhelio_row_col_length.y / 2, 0.0f, subhelio_row_col_length.x / 2));
    localSubVertexes.push_back(make_float3(-subhelio_row_col_length.y / 2, 0.0f, -subhelio_row_col_length.x / 2));
    localSubVertexes.push_back(make_float3(subhelio_row_col_length.y / 2, 0.0f, -subhelio_row_col_length.x / 2));

    float3 *h_local_centers = new float3[row_col_.x * row_col_.y];
    float3 *h_local_normals = new float3[row_col_.x * row_col_.y];

    for (int r = 0; r < row_col_.x; ++r) {
        for (int c = 0; c < row_col_.y; ++c) {
            int id = global_func::unroll_index(make_int2(r, c), row_col_);

            // local centers
            h_local_centers[id] = make_float3(
                    c * (gap_.x + subhelio_row_col_length.y) - size_.x / 2 + subhelio_row_col_length.y / 2,
                    size_.y / 2,
                    r * (gap_.y + subhelio_row_col_length.x) - size_.z / 2 + subhelio_row_col_length.x / 2);
            h_local_centers[id].y += (h_local_centers[id].x * h_local_centers[id].x +
                                      h_local_centers[id].z * h_local_centers[id].z) / (4 * focus_length_);

            // local normals
            h_local_normals[id] = make_float3(-1 / (2 * focus_length_) * h_local_centers[id].x,
                                              1.0f,
                                              -1 / (2 * focus_length_) * h_local_centers[id].z);
            h_local_normals[id] = normalize(h_local_normals[id]);

            for (float3 subHeliostatVertex : localSubVertexes) {
                subHeliostatVertex = focusFlatRectangle_heliostat::focusFlatRectangleHeliostatLocal2World(
                        subHeliostatVertex, h_local_normals[id]);
                subHeliostatVertex = global_func::transform(subHeliostatVertex, h_local_centers[id]);

                subHeliostatVertex = focusFlatRectangle_heliostat::focusFlatRectangleHeliostatLocal2World(
                        subHeliostatVertex, normal_);
                subHeliostatVertex = global_func::transform(subHeliostatVertex, pos_);

                subHeliostatVertexes.push_back(subHeliostatVertex);
            }
        }
    }

    // Copy the local centers and normals from CPU to GPU
    global_func::cpu2gpu(d_local_centers, h_local_centers, row_col_.x * row_col_.y);
    global_func::cpu2gpu(d_local_normals, h_local_normals, row_col_.x * row_col_.y);

    // Clear
    delete[] h_local_centers;
    delete[] h_local_normals;
    h_local_centers = nullptr;
    h_local_normals = nullptr;
}

float FocusFlatRectangleHelio::getFocusLength() const {
    return focus_length_;
}

void FocusFlatRectangleHelio::setFocusLength(float focus_length) {
    focus_length_ = focus_length;
}

std::vector<float3> FocusFlatRectangleHelio::getGPULocalNormals() {
    float3 *h_local_normals = new float3[row_col_.x * row_col_.y];
    global_func::gpu2cpu(h_local_normals, d_local_normals, row_col_.x * row_col_.y);

    std::vector<float3> cpu_local_normals;
    for (int i = 0; i < row_col_.x * row_col_.y; ++i) {
        cpu_local_normals.push_back(h_local_normals[i]);
    }

    // Clear
    delete[] h_local_normals;
    h_local_normals = nullptr;

    return cpu_local_normals;
}

void FocusFlatRectangleHelio::setGPULocalNormals(float3 *h_local_normals) {
    global_func::cpu2gpu(d_local_normals, h_local_normals, row_col_.x * row_col_.y);
}

void FocusFlatRectangleHelio::setGPULocalNormals(std::vector<float3> local_normals) {
    float3 *h_local_normals = new float3[row_col_.x * row_col_.y];
    for (int i = 0; i < row_col_.x * row_col_.y; ++i) {
        h_local_normals[i] = local_normals[i];
    }
    global_func::cpu2gpu(d_local_normals, h_local_normals, row_col_.x * row_col_.y);

    delete[] h_local_normals;
    h_local_normals = nullptr;
}

std::vector<float3> FocusFlatRectangleHelio::getGPULocalCenters() {
    float3 *h_local_centers = new float3[row_col_.x * row_col_.y];
    global_func::gpu2cpu(h_local_centers, d_local_centers, row_col_.x * row_col_.y);

    std::vector<float3> cpu_local_centers;
    for (int i = 0; i < row_col_.x * row_col_.y; ++i) {
        cpu_local_centers.push_back(h_local_centers[i]);
    }

    // Clear
    delete[] h_local_centers;
    h_local_centers = nullptr;

    return cpu_local_centers;
}

void FocusFlatRectangleHelio::setGPUocalCenters(float3 *h_local_centers) {
    global_func::cpu2gpu(d_local_centers, h_local_centers, row_col_.x * row_col_.y);
}

void FocusFlatRectangleHelio::setGPUocalCenters(std::vector<float3> local_centers) {
    float3 *h_local_centers = new float3[row_col_.x * row_col_.y];
    for (int i = 0; i < row_col_.x * row_col_.y; ++i) {
        h_local_centers[i] = local_centers[i];
    }
    global_func::cpu2gpu(d_local_centers, h_local_centers, row_col_.x * row_col_.y);

    delete[] h_local_centers;
    h_local_centers = nullptr;
}

void FocusFlatRectangleHelio::setSurfaceProperty(const std::vector<float> &surface_property) {
    if (surface_property.empty()) {
        return;
    }

    focus_length_ = surface_property[0];
}

std::vector<float> FocusFlatRectangleHelio::getSurfaceProperty() {
    std::vector<float> ans(6, -1.0f);
    ans[0] = focus_length_;
    return ans;
}

void FocusFlatRectangleHelio::CSetNormalAndRotate(const float3 &focus_center, const float3 &sunray_dir) {
    if (focus_length_ < 0.0f) {
        focus_length_ = length(focus_center - pos_);
    }
    Heliostat::CSetNormalAndRotate(focus_center, sunray_dir);
}
