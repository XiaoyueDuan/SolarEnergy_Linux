#include "hip/hip_runtime.h"
#include "Heliostat.cuh"
#include "vector_arithmetic.cuh"
#include "global_function.cuh"

void Heliostat::CSetNormalAndRotate(const float3 &focus_center, const float3 &sunray_dir) {
    CSetNormal(focus_center, sunray_dir);
    CSetWorldVertex();
}

float3 Heliostat::getPosition() const {
    return pos_;
}

void Heliostat::setPosition(float3 pos) {
    pos_ = pos;
}

float3 Heliostat::getSize() const {
    return size_;
}

void Heliostat::setSize(float3 size) {
    size_ = size;
}

float3 Heliostat::getNormal() const {
    return normal_;
}

void Heliostat::setNormal(float3 normal) {
    normal_ = normal;
}

int2 Heliostat::getRowAndColumn() const {
    return row_col_;
}

void Heliostat::setRowAndColumn(int2 row_col) {
    row_col_ = row_col;
}

float2 Heliostat::getGap() const {
    return gap_;
}

void Heliostat::setGap(float2 gap) {
    gap_ = gap;
}

SubCenterType Heliostat::getSubCenterType() const {
    return subCenterType_;
}

void Heliostat::setSubCenterType(SubCenterType type) {
    subCenterType_ = type;
}

float Heliostat::getPixelLength() const {
    return pixel_length_;
}

void Heliostat::setPixelLength(float pixel_length) {
    pixel_length_ = pixel_length;
}

void Heliostat::Cget_vertex(float3 &v0, float3 &v1, float3 &v3) {
    v0 = vertex_[0];
    v1 = vertex_[1];
    v3 = vertex_[3];
}

void Heliostat::CSetWorldVertex() {
    vertex_[0] = make_float3(-size_.x / 2, size_.y / 2, -size_.z / 2);
    vertex_[1] = vertex_[0] + make_float3(0, 0, size_.z);
    vertex_[2] = vertex_[0] + make_float3(size_.x, 0, size_.z);
    vertex_[3] = vertex_[0] + make_float3(size_.x, 0, 0);

    vertex_[0] = global_func::local2world(vertex_[0], normal_);
    vertex_[1] = global_func::local2world(vertex_[1], normal_);
    vertex_[2] = global_func::local2world(vertex_[2], normal_);
    vertex_[3] = global_func::local2world(vertex_[3], normal_);

    vertex_[0] = global_func::transform(vertex_[0], pos_);
    vertex_[1] = global_func::transform(vertex_[1], pos_);
    vertex_[2] = global_func::transform(vertex_[2], pos_);
    vertex_[3] = global_func::transform(vertex_[3], pos_);
}

void Heliostat::CSetNormal(const float3 &focus_center, const float3 &sunray_dir) {
    float3 local_center = make_float3(pos_.x, pos_.y, pos_.z);
    float3 reflect_dir = focus_center - local_center;
    reflect_dir = normalize(reflect_dir);
    float3 dir = reflect_dir - sunray_dir;
    normal_ = normalize(dir);
}
