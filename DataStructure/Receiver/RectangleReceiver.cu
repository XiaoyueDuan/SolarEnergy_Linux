#include "hip/hip_runtime.h"
#include "RectangleReceiver.cuh"
#include "global_function.cuh"

bool RectangleReceiver::GIntersect(const float3 &orig, const float3 &dir,
        float &t, float &u, float &v)
{
    return global_func::rayParallelogramIntersect(orig, dir,
            rect_vertex_[0], rect_vertex_[1], rect_vertex_[3], t, u, v);
}

// RectangleReceiver
void RectangleReceiver::CInit(int geometry_info)
{
    pixel_length_ = 1.0f / float(geometry_info);
    Cinit_vertex();
    Cset_focuscenter();
    Cset_resolution(geometry_info);
    Calloc_image();
    Cclean_image_content();
}

void RectangleReceiver::Cinit_vertex()
{
    Cset_localnormal();	// set local normal
    Cset_localvertex();	// set local vertex according to face type
    Cset_vertex();		// set world vertex according to normal
}

void RectangleReceiver::Cset_resolution(int geometry_info)
{
    resolution_.x = size_.x*float(geometry_info);
    resolution_.y = size_.y*float(geometry_info);
}

void RectangleReceiver::Cset_focuscenter()
{
    focus_center_ = (rect_vertex_[0] + rect_vertex_[2]) / 2;
}

void RectangleReceiver::Cset_localnormal()
{
    switch (face_num_)
    {
        case 0:
            localnormal_ = make_float3(0.0f, 0.0f, 1.0f);
            break;
        case 1:
            localnormal_ = make_float3(1.0f, 0.0f, 0.0f);
            break;
        case 2:
            localnormal_ = make_float3(0.0f, 0.0f, -1.0f);
            break;
        case 3:
            localnormal_ = make_float3(-1.0f, 0.0f, 0.0f);
            break;
        default:
            break;
    }
}

void RectangleReceiver::Cset_localvertex()
{
    switch (face_num_)
    {
        case 0:
            rect_vertex_[0] = make_float3(-size_.x / 2, -size_.y / 2, size_.z / 2);
            rect_vertex_[1] = make_float3(-size_.x / 2, size_.y / 2, size_.z / 2);
            rect_vertex_[2] = make_float3(size_.x / 2, size_.y / 2, size_.z / 2);
            rect_vertex_[3] = make_float3(size_.x / 2, -size_.y / 2, size_.z / 2);
            break;
        case 1:
            rect_vertex_[0] = make_float3(size_.x / 2, -size_.y / 2, size_.z / 2);
            rect_vertex_[1] = make_float3(size_.x / 2, size_.y / 2, size_.z / 2);
            rect_vertex_[2] = make_float3(size_.x / 2, size_.y / 2, -size_.z / 2);
            rect_vertex_[3] = make_float3(size_.x / 2, -size_.y / 2, -size_.z / 2);
            break;
        case 2:
            rect_vertex_[0] = make_float3(size_.x / 2, -size_.y / 2, -size_.z / 2);
            rect_vertex_[1] = make_float3(size_.x / 2, size_.y / 2, -size_.z / 2);
            rect_vertex_[2] = make_float3(-size_.x / 2, size_.y / 2, -size_.z / 2);
            rect_vertex_[3] = make_float3(-size_.x / 2, -size_.y / 2, -size_.z / 2);
            break;
        case 3:
            rect_vertex_[0] = make_float3(-size_.x / 2, -size_.y / 2, -size_.z / 2);
            rect_vertex_[1] = make_float3(-size_.x / 2, size_.y / 2, -size_.z / 2);
            rect_vertex_[2] = make_float3(-size_.x / 2, size_.y / 2, size_.z / 2);
            rect_vertex_[3] = make_float3(-size_.x / 2, -size_.y / 2, size_.z / 2);
            break;
        default:
            break;
    }
}

void RectangleReceiver::Cset_vertex()
{
    normal_ = normalize(normal_);
    rect_vertex_[0] = global_func::rotateY(rect_vertex_[0], localnormal_, normal_);
    rect_vertex_[1] = global_func::rotateY(rect_vertex_[1], localnormal_, normal_);
    rect_vertex_[2] = global_func::rotateY(rect_vertex_[2], localnormal_, normal_);
    rect_vertex_[3] = global_func::rotateY(rect_vertex_[3], localnormal_, normal_);

    rect_vertex_[0] = global_func::transform(rect_vertex_[0], pos_);
    rect_vertex_[1] = global_func::transform(rect_vertex_[1], pos_);
    rect_vertex_[2] = global_func::transform(rect_vertex_[2], pos_);
    rect_vertex_[3] = global_func::transform(rect_vertex_[3], pos_);
}